
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4
#define THREADS_PER_BLOCK 256

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // Initialize input vectors a and b
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Calculate the number of blocks needed
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Perform vector addition on GPU
    vectorAdd<<<numBlocks, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c, N);

    // Copy result back to host
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Display the result
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
